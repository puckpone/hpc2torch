#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <algorithm>    // 用于std::min

template <typename T, typename Tind>
__global__ void blockGatherKernel(T const *input, Tind const *indices, T *output, int stride, int indSize)
{
    int tid = blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) * indSize; 
    
    // 添加循环展开指令（修正语法）
    #pragma unroll 4
    for (int index = threadIdx.x; index < indSize; index += blockDim.x) {
        const Tind src_offset = indices[index] * stride;
        output[tid + index * stride] = input[tid + src_offset];
    }
}

template <typename T, typename Tind>
__global__ void warpGatherKernel(T const *input, Tind const *indices, T *output, int stride, int indSize)
{
    int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
    int tid = otherIdx % stride + (otherIdx - otherIdx % stride) * indSize;
    
    // 添加循环展开指令（修正语法）
    #pragma unroll 4
    for (int index = threadIdx.x; index < indSize; index += blockDim.x) {
        const Tind src_offset = indices[index] * stride;
        output[tid + index * stride] = input[tid + src_offset];
    }
}

template <typename T, typename Tind>
void gatherLaunch(void const *input, void const *indices, void *output, int stride, int indSize, int othersize)
{
    if (indSize > 1024)
    {
        // 优化线程块配置（保证warp对齐）
        int blockDim = (stride % 32 == 0) ? std::min(1024, stride) : ((stride/32)+1)*32;
        blockDim = std::min(blockDim, 1024);
        blockGatherKernel<T, Tind>
            <<<othersize, blockDim>>>((T *)input, (Tind *)indices, (T *)output, stride, indSize);
    }
    else if (indSize > 31)
    {
        dim3 block_dim(32, 32);
        dim3 grid_dim((othersize + 31)/32, 1);
        warpGatherKernel<T, Tind>
            <<<grid_dim, block_dim>>>((T *)input, (Tind *)indices, (T *)output, stride, indSize);
    }
    else if (indSize > 15)
    {
        int BLOCK_DIM_y = std::min(256, othersize);
        dim3 block_dim(16, BLOCK_DIM_y);
        dim3 grid_dim(1);
        warpGatherKernel<T, Tind>
            <<<grid_dim, block_dim>>>((T *)input, (Tind *)indices, (T *)output, stride, indSize);
    }
    else
    {
        int BLOCK_DIM_y = std::min(256, othersize);
        dim3 block_dim(4, BLOCK_DIM_y);
        dim3 grid_dim(1);
        warpGatherKernel<T, Tind>
            <<<grid_dim, block_dim>>>((T *)input, (Tind *)indices, (T *)output, stride, indSize);
    }
}

extern "C" void gather_nv_f32(void const *input, void const *indices, void *output, int stride, int indSize, int othersize) {
    gatherLaunch<float, uint64_t>(input, indices, output, stride, indSize, othersize);
}

extern "C" void gather_nv_f16(void const *input, void const *indices, void *output, int stride, int indSize, int othersize) {
    gatherLaunch<half, uint64_t>(input, indices, output, stride, indSize, othersize);
}